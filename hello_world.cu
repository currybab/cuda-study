#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void helloCUDA(void) {
    printf("Hello CUDA from GPU!\n");
}

int main(void) {
    printf("Hello GPU from CPU!\n");
    helloCUDA<<<1,10>>>();
    hipDeviceSynchronize();
    return 0;
}